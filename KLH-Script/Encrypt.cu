#include "hip/hip_runtime.h"


%%writefile password_cracker.cu
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__device__ char* CudaCrypt(char* rawPassword) {
    char *newPassword = (char *) malloc(sizeof(char) * 11);

    newPassword[0] = rawPassword[0] + 2;
    newPassword[1] = rawPassword[0] - 2;
    newPassword[2] = rawPassword[0] + 1;
    newPassword[3] = rawPassword[1] + 3;
    newPassword[4] = rawPassword[1] - 3;
    newPassword[5] = rawPassword[1] - 1;
    newPassword[6] = rawPassword[2] + 2;
    newPassword[7] = rawPassword[2] - 2;
    newPassword[8] = rawPassword[3] + 4;
    newPassword[9] = rawPassword[3] - 4;
    newPassword[10] = '\0';

    for (int i = 0; i < 10; i++) {
        if (i >= 0 && i < 6) { // checking all uppercase letter limits
            if (newPassword[i] > 90) {
                newPassword[i] = (newPassword[i] - 90) + 65;
            } else if (newPassword[i] < 65) {
                newPassword[i] = (65 - newPassword[i]) + 65;
            }
        } else { // checking number section
            if (newPassword[i] > 57) {
                newPassword[i] = (newPassword[i] - 57) + 48;
            } else if (newPassword[i] < 48) {
                newPassword[i] = (48 - newPassword[i]) + 48;
            }
        }
    }
    return newPassword;
}

__global__ void crack(char *alphabet, char *numbers, char *userPassword, unsigned long long *combinationCount, unsigned long long *lastThreadId) {
    char genRawPass[4];

    // Generating password characters
    genRawPass[0] = alphabet[blockIdx.x];
    genRawPass[1] = alphabet[blockIdx.y];
    genRawPass[2] = numbers[threadIdx.x];
    genRawPass[3] = numbers[threadIdx.y];

    // Calculate unique thread ID
    unsigned long long threadId = (blockIdx.x * gridDim.y * blockDim.x * blockDim.y) +
                                  (blockIdx.y * blockDim.x * blockDim.y) +
                                  (threadIdx.x * blockDim.y) +
                                  threadIdx.y;

    // Compare the generated password with the user input
    bool match = true;
    for (int i = 0; i < 4; i++) {
        if (genRawPass[i] != userPassword[i]) {
            match = false;
            break;
        }
    }

    if (match) {
        // If the password matches, print the thread ID and the generated password
        printf("Thread ID: %llu | %c %c %c %c = %s\n", threadId, genRawPass[0], genRawPass[1], genRawPass[2], genRawPass[3], CudaCrypt(genRawPass));
    }

    // Update combination count
    atomicAdd(combinationCount, 1ULL);

    // Record the last thread ID
    atomicMax(lastThreadId, threadId);
}

int main(int argc, char **argv) {
    char cpuAlphabet[26] = {'A','B','C','D','E','F','G','H','I','J','K','L','M','N','O','P','Q','R','S','T','U','V','W','X','Y','Z'};
    char cpuNumbers[10] = {'0','1','2','3','4','5','6','7','8','9'};

    char *gpuAlphabet;
    hipMalloc((void**) &gpuAlphabet, sizeof(char) * 26);
    hipMemcpy(gpuAlphabet, cpuAlphabet, sizeof(char) * 26, hipMemcpyHostToDevice);

    char *gpuNumbers;
    hipMalloc((void**) &gpuNumbers, sizeof(char) * 10);
    hipMemcpy(gpuNumbers, cpuNumbers, sizeof(char) * 10, hipMemcpyHostToDevice);

    // Device variables for tracking total combinations and last thread ID
    unsigned long long *d_combinationCount, *d_lastThreadId;
    unsigned long long h_combinationCount = 0, h_lastThreadId = 0;
    hipMalloc((void**) &d_combinationCount, sizeof(unsigned long long));
    hipMalloc((void**) &d_lastThreadId, sizeof(unsigned long long));
    hipMemcpy(d_combinationCount, &h_combinationCount, sizeof(unsigned long long), hipMemcpyHostToDevice);
    hipMemcpy(d_lastThreadId, &h_lastThreadId, sizeof(unsigned long long), hipMemcpyHostToDevice);

    // Prompt user for a password to search for
    char userPassword[5];
    printf("Enter a 4-character password to search for: ");
    scanf("%4s", userPassword);

    // Allocate memory for user password on the GPU
    char *gpuUserPassword;
    hipMalloc((void**)&gpuUserPassword, sizeof(char) * 5);
    hipMemcpy(gpuUserPassword, userPassword, sizeof(char) * 5, hipMemcpyHostToDevice);

    // Launch the kernel
    dim3 grid(26, 26, 1);
    dim3 block(10, 10, 1);
    crack<<<grid, block>>>(gpuAlphabet, gpuNumbers, gpuUserPassword, d_combinationCount, d_lastThreadId);
    hipDeviceSynchronize();

    // Copy results back to host
    hipMemcpy(&h_combinationCount, d_combinationCount, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    hipMemcpy(&h_lastThreadId, d_lastThreadId, sizeof(unsigned long long), hipMemcpyDeviceToHost);

    // Print summary
    printf("Total number of passwords explored: %llu\n", h_combinationCount);
    printf("Last thread ID: %llu\n", h_lastThreadId);

    // Cleanup
    hipFree(gpuAlphabet);
    hipFree(gpuNumbers);
    hipFree(gpuUserPassword);
    hipFree(d_combinationCount);
    hipFree(d_lastThreadId);

    return 0;
}


